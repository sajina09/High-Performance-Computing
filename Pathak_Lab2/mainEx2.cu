
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>

#define cudaCheckError() {                                                                       \
        hipError_t e=hipGetLastError();                                                        \
        if(e!=hipSuccess) {                                                                     \
            printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));        \
            exit(EXIT_FAILURE);                                                                  \
        }                                                                                        \
    }

__global__ void kernel(int *a, int N) {
  int i=blockIdx.x*blockDim.x+threadIdx.x;
  // Check to prevent out-of-bounds memory access
    if (i < N) {
        a[i] = i;
    }
}

int main() {
    printf("Main starts here.");

  
  int N=4097;
  int threads=128;
  int blocks=(N+threads-1)/threads;
  int *a;

  hipMallocManaged(&a,N*sizeof(int));
  kernel<<<blocks,threads>>>(a, N);
  hipDeviceSynchronize();

  for(int i=0;i<10;i++)
    printf("%d\n",a[i]);

  hipFree(a);

  cudaCheckError();
  return 0;
}


